#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "helpers.h"
#include "path_tracer.h"
#include "random.h"
#include "utils.h"

using namespace optix;

struct PerRayData_radiance{
    float3 result;
    float importance;
    int depth;
};


struct PerRayData_pathtrace_shadow{
    bool inShadow;
};

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;


rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, current_prd, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

static __device__ inline float3 powf(float3 a, float exp)
{
  return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
}

// For miss program
rtDeclareVariable(float3,       bg_color, , );
rtDeclareVariable(float3,        emission_color, , );

// For envirnoment map
rtTextureSampler<float4, 2> envmap;

// For shadow ray
rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

// For vertex tracer
rtBuffer<MyVertex>  vertices;

// For diffuse texture map
rtTextureSampler<float4, 2>   diffuse_map;         

RT_PROGRAM void exception(){
  output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


RT_PROGRAM void miss(){
  current_prd.result = bg_color;
}


RT_PROGRAM void shadow(){
  current_prd_shadow.inShadow = true;
  rtTerminateRay();
}


RT_PROGRAM void envmap_miss(){
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	current_prd.result = make_float3(tex2D(envmap, u, v));
}


RT_PROGRAM void one_bounce_diffuse_closest_hit(){
    float3 hit_point = ray.origin + t_hit * ray.direction;
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal               = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    float2 uv                     = make_float2(texcoord);

    float3 Kd = make_float3(tex2D(diffuse_map, uv.x, uv.y));
    float3 result = make_float3(0);

    // compute indirect bounce 
    if(prd.detph < 1){
        optix::Onb onb(ffnormal);
        unsigned int seed = rot_seed(rnd_seeds[launch_index], frame);
        const float inv_sqrt_samples = 1.0f / float(sqrt_num_samples);

        int nx = sqrt_num_samples;
        int ny = sqrt_num_samples;

        while(ny--){
            while(nx--){
                float u1 = (float(nx) + rnd(seed)) * inv_sqrt_samples;
                float u2 = (float(ny) + rnd(seed)) * inv_sqrt_samples;

                float3 dir;
                optix::cosine_sample_hemisphere(u1, u2, dir);
                onb.inverse_transform(dir);

                PerRayData_radiance radiance_prd;
                radiance_prd.importance = current_prd.importance * optix::luminance(Kd);
                radiance_prd.depth = current_prd.depth + 1;

                if(radiance_prd.importance > 0.001f){
                    optix::Ray radiance_ray = optix::make_Ray(hit_point, dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
                    rtTrace(top_object, radiance_ray, radiance_prd);
                    result += radiance_prd.result;
                }
            }
            nx = sqrt_num_samples;
        }
        result *= (Kd) / ((float)(sqrt_num_samples * sqrt_num_samples));
    }

    current_prd.result = result;
}


RT_PROGRAM void vertex_camera(){
    float3 vertex_pos = vertices[launch_index.x].vertex;
    float3 vertex_normal = vertices[launch_index.x].normal;
    vertex_normal = normalize(vertex_normal);
    float3 result = make_float3(0);

    const float inv_sqrt_samples = 1.0f / (sqrt_num_samples);
    int nx = sqrt_num_samples;
    int ny = sqrt_num_samples;
    unsigned int seed = rot_seed( rnd_seeds[ launch_index ], frame );

    optix::Onb onb(vertex_normal);
    float3 Kd = make_float3(1.0, 1.0, 1.0);
    while(ny--){
        while(nx--){
            float u1 = (float(nx) + rnd( seed ) )*inv_sqrt_samples;
            float u2 = (float(ny) + rnd( seed ) )*inv_sqrt_samples;

            float3 dir;
            optix::cosine_sample_hemisphere(u1, u2, dir);
            onb.inverse_transform(dir);

            PerRayData_radiance radiance_prd;
            radiance_prd.importance = optix::luminance(Kd);
            radiance_prd.depth = 0;
            if(radiance_prd.importance > 0.001f) {
                optix::Ray radiance_ray = optix::make_Ray(hit_point, dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
                rtTrace(top_object, radiance_ray, radiance_prd);

                result += radiance_prd.result;
            }
        }
        nx = sqrt_num_samples;
    }

    result *= (Kd)/((float)(sqrt_diffuse_samples*sqrt_diffuse_samples));

    output_buffer[launch_index] = make_float4(result, 0.0f);
}





