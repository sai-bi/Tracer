#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "helpers.h"
#include "path_tracer.h"
#include "random.h"
#include "utils.h"

using namespace optix;

struct PerRayData_pathtrace
{
  float3 result;
  float3 radiance;
  float3 attenuation;
  float3 origin;
  float3 direction;
  unsigned int seed;
  int depth;
  int countEmitted;
  int done;
  int inside;

  // @sai bi
  Matrix3x3 sh_coeff;
  Matrix3x3 sh_result;
};

struct PerRayData_pathtrace_shadow
{
  bool inShadow;
};

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;

rtBuffer<float4, 2> output_buffer_1;
rtBuffer<float4, 2> output_buffer_2;
rtBuffer<float4, 2> output_buffer_3;


rtBuffer<ParallelogramLight>     lights;

rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

static __device__ inline float3 powf(float3 a, float exp)
{
  return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
}

// For miss program
rtDeclareVariable(float3,       bg_color, , );

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
  current_prd.radiance = current_prd.countEmitted? emission_color : make_float3(0.f);
  current_prd.done = true;
}

// rtDeclareVariable(float3, diffuse_color, , );
rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void diffuse()
{
  // if(current_prd.depth == 1){
	 //  current_prd.sh_coeff = 0.0 * current_prd.sh_coeff;
  //     return;
  // }

  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

  float3 hitpoint = ray.origin + t_hit * ray.direction;
  current_prd.origin = hitpoint;

  // by bisai 
  float3 diffuse_color = make_float3(0.8f, 0.8f, 0.8f);

  float z1=rnd(current_prd.seed);
  float z2=rnd(current_prd.seed);
  float3 p;
  cosine_sample_hemisphere(z1, z2, p);
  float3 v1, v2;
  createONB(ffnormal, v1, v2);
  current_prd.direction = v1 * p.x + v2 * p.y + ffnormal * p.z;

  // float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;
  // current_prd.attenuation = current_prd.attenuation * diffuse_color; // use the diffuse_color as the diffuse response
   current_prd.countEmitted = false;
   current_prd.sh_coeff = 0.0 * current_prd.sh_coeff;

   Matrix3x3 li;
   li  = 0.0 * li;
   //return;
   int resolution = 10;
   for(int i = 0;i < resolution;i++){
       for(int j = 0;j < resolution;j++){
            float z1 = (i + 0.5) / (float)resolution;
            float z2 = (j + 0.5)/ (float)resolution;
            float3 p;
            cosine_sample_hemisphere(z1, z2, p);
            float3 ray_direction = v1 * p.x  + v2 * p.y  + ffnormal * p.z;

			PerRayData_pathtrace_shadow shadow_prd;
			Ray shadow_ray = make_Ray(hitpoint, ray_direction, pathtrace_shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX);
		    shadow_prd.inShadow = false;
			rtTrace(top_object, shadow_ray, shadow_prd);

			float3 d = ray_direction;
			Matrix3x3 result;
			float* data = result.getData();
			if (!shadow_prd.inShadow){
				data[0] = 0.282095;
				data[1] = 0.488603 * d.y; data[2] = 0.488603 * d.z; data[3] = 0.488603 * d.x;
				data[4] = 1.092548 * d.x * d.y;
				data[5] = 1.092548 * d.y * d.z;
				data[6] = 0.315392 * (3 * d.z * d.z - 1);
				data[7] = 1.092548 * d.x * d.z;
				data[8] = 0.546274 * (d.x * d.x - d.y * d.y);
				/// li += (result * (0.7 / M_PIf));
				li += result;
			}
       }
   }

   current_prd.sh_coeff = li / (resolution * resolution);

  // current_prd.sh_result = li / (resolution * resolution); 

  // // Compute direct light...
  // // Or shoot one...
  // float3 result = make_float3(0.0f);

  // sample direct illumination
  /*
  unsigned int sample_num = 400;
  for (int i = 0;i < sample_num;i++){
      // float z1=rnd(current_prd.seed);
      // float z2=rnd(current_prd.seed);
      float z1 = (i % 20 + 0.5) / 20;
      float z2 = (i / 20 + 0.5) / 20;
      float3 p;
      cosine_sample_hemisphere(z1, z2, p);
      float3 direction = normalize(v1 * p.x + v2 * p.y + ffnormal * p.z);

      PerRayData_pathtrace_shadow shadow_prd;
      Ray shadow_ray = make_Ray( hitpoint, direction, pathtrace_shadow_ray_type, scene_epsilon, 100);
      rtTrace(top_object, shadow_ray, shadow_prd);
      
      if(!shadow_prd.inShadow){
          float theta = atan2f(direction.x, direction.z);
          float phi = M_PIf * 0.5f - acosf(direction.y);
          float u = (theta + M_PIf) * (0.5f * M_1_PIf);
          float v = 0.5f * (1.0f + sin(phi));
          float3 color = make_float3(tex2D(envmap, u, v));
          result = result + color;
      }
  }

  result = result / sample_num;
  */
  // compute light from environment map
  // current_prd.radiance = result;
  // current_prd.sh_coeff = 0.0 * current_prd.sh_coeff;
}

rtDeclareVariable(float3,        glass_color, , );
rtDeclareVariable(float,         index_of_refraction, , );

RT_PROGRAM void glass_refract()
{
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

  float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hitpoint = ray.origin + t_hit * ray.direction;
  current_prd.origin = hitpoint;
  current_prd.countEmitted = true;
  float iof;
  if (current_prd.inside) {
    // Shoot outgoing ray
    iof = 1.0f/index_of_refraction;
  } else {
    iof = index_of_refraction;
  }
  refract(current_prd.direction, ray.direction, ffnormal, iof);
  //prd.direction = reflect(ray.direction, ffnormal);

  if (current_prd.inside) {
    // Compute Beer's law
    current_prd.attenuation = current_prd.attenuation * powf(glass_color, t_hit);
  }
  current_prd.inside = !current_prd.inside;

  current_prd.radiance = make_float3(0.0f);
}

//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void miss()
{
  current_prd.radiance = bg_color;
  current_prd.done = true;
}


rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );
RT_PROGRAM void shadow()
{
  current_prd_shadow.inShadow = true;
  rtTerminateRay();
}


RT_PROGRAM void envmap_miss()
{
    // printf("envmap miss\n");
    // float theta = atan2f(ray.direction.x, ray.direction.z);
    // float phi = M_PIf * 0.5f - acosf(ray.direction.y);
    // float u = (theta + M_PIf) * (0.5f * M_1_PIf);
    // float v = 0.5f * (1.0f + sin(phi));
    // current_prd.radiance = make_float3(tex2D(envmap, u, v));
    float3 d = normalize(ray.direction);
    
    Matrix3x3 result;
    float* data = result.getData();
    data[0] = 0.282095;
    data[1] = 0.488603 * d.y; data[2] = 0.488603 * d.z; data[3] = 0.488603 * d.x;
    data[4] = 1.092548 * d.x * d.y;
    data[5] = 1.092548 * d.y * d.z;
    data[6] = 0.315392 * (3 * d.z * d.z - 1);
    data[7] = 1.092548 * d.x * d.z;
    data[8] = 0.546274 * (d.x * d.x - d.y * d.y);
    
    current_prd.sh_coeff = result;
    current_prd.done = true;
}


// Pure mirrow reflection
RT_PROGRAM void MirrorReflection(){
    float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));

    float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

    float3 hitpoint = ray.origin + t_hit * ray.direction;
    current_prd.origin = hitpoint;

    // by bisai 
    float3 diffuse_color = make_float3(0.8f, 0.8f, 0.8f);

    float z1 = rnd(current_prd.seed);
    float z2 = rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    float3 v1, v2;
    createONB(ffnormal, v1, v2);
    
    float3 reflect_direction = reflect(ray.direction, ffnormal);
    current_prd.direction = reflect_direction; // specular reflection

    float3 normal_color = (normalize(world_shading_normal)*0.5f + 0.5f)*0.9;
    current_prd.attenuation = current_prd.attenuation * diffuse_color; // use the diffuse_color as the diffuse response
    current_prd.countEmitted = false;

    current_prd.radiance = make_float3(0.0f);
}

rtBuffer<MyVertex>  vertices;
rtDeclareVariable(unsigned int, samples_per_vertex, , );
// ray trace each vertex
RT_PROGRAM void VertexTracer(){
    int index = launch_index.x;
    float3 vertex_pos = vertices[index].vertex;
    float3 normal = vertices[index].normal;
    normal = normalize(normal);

    unsigned int seed = tea<16>(index, 1);
    unsigned int sample_num = samples_per_vertex;
    float3 result = make_float3(0.0f);

    float3 v1, v2;
    createONB(normal, v1, v2);
    float3 direct_light = make_float3(0.0);
    sample_num = 1000;


    do{
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.0f);
        prd.attenuation = make_float3(0.8f);
        prd.countEmitted = true;
        prd.done = false;
        prd.inside = false;
        prd.seed = seed;
        prd.depth = 0;

        // float z1 = (sample_num % 20 + 0.5) / 20;
        // float z2 = (sample_num / 20 + 0.5) / 20;
        float z1 = rnd(prd.seed);
        float z2 = rnd(prd.seed);

        float3 p;
        cosine_sample_hemisphere(z1, z2, p);
        float3 ray_direction = v1 * p.x  + v2 * p.y  + normal * p.z;

        ray_direction = normalize(ray_direction);
        float cos_theta = dot(ray_direction, normal);

        float3 ray_origin = vertex_pos;
        for(;;) {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);
            if(prd.done) {
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // RR: randomly reject some rays
            if(prd.depth >= rr_begin_depth){
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += (prd.result);
        seed = prd.seed;
    } while(--sample_num);


    result = result / 1000;

    float3 pixel_color = result;
    output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
}



RT_PROGRAM void DirectRender(){
    unsigned int resolution = 64;
    int index = launch_index.x;
    float3 vertex_pos = vertices[index].vertex;
    float3 normal = vertices[index].normal;
    normal = normalize(normal);
    unsigned int seed = tea<16>(index, 1);

    float3 v1, v2;
    createONB(normal, v1, v2);
    Matrix3x3 li; 
    li = 0.0 * li;

    for(unsigned int i = 0;i < resolution;i++){
        for (unsigned int j = 0;j < resolution; j++){
            float z1 = (i + 0.5) / (float)resolution;
            float z2 = (j + 0.5)/ (float)resolution;
            float3 p;
            cosine_sample_hemisphere(z1, z2, p);
            float3 ray_direction = v1 * p.x  + v2 * p.y  + normal * p.z;
                        
            ray_direction = normalize(ray_direction);

            PerRayData_pathtrace_shadow shadow_prd;
            Ray shadow_ray = make_Ray(vertex_pos, ray_direction, pathtrace_shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            shadow_prd.inShadow = false;
            rtTrace(top_object, shadow_ray, shadow_prd);

            float3 d = ray_direction;
            Matrix3x3 result;
            float* data = result.getData();
            if(!shadow_prd.inShadow){
                data[0] = 0.282095;
                data[1] = 0.488603 * d.y; 
				data[2] = 0.488603 * d.z; 
				data[3] = 0.488603 * d.x;
                data[4] = 1.092548 * d.x * d.y;
                data[5] = 1.092548 * d.y * d.z;
                data[6] = 0.315392 * (3 * d.z * d.z - 1);
                data[7] = 1.092548 * d.x * d.z;
                data[8] = 0.546274 * (d.x * d.x - d.y * d.y);
                /// li += (result * (0.7 / M_PIf));
                li += result;
            }
        }
    }
    li = li / (resolution * resolution);
    // li = li * (2 * M_PIf / (resolution * resolution));

    output_buffer_1[launch_index] = make_float4(li[0], li[1], li[2], 0.0f);
    output_buffer_2[launch_index] = make_float4(li[3], li[4], li[5], 0.0f);
    output_buffer_3[launch_index] = make_float4(li[6], li[7], li[8], 0.0f);
}

RT_PROGRAM void InDirectRender(){
    unsigned int resolution = 10;
    int index = launch_index.x;
    float3 vertex_pos = vertices[index].vertex;
    float3 normal = vertices[index].normal;
    normal = normalize(normal);
    unsigned int seed = tea<16>(index, 1);

    float3 v1, v2;
    createONB(normal, v1, v2);
    Matrix3x3 li;
    li = 0.0 * li;

    for (unsigned int i = 0; i < resolution; i++){
        for (unsigned int j = 0; j < resolution; j++){
            float z1 = (i + 0.5) / (float)resolution;
            float z2 = (j + 0.5) / (float)resolution;
            
            PerRayData_pathtrace prd;
            prd.sh_result = 0.0f * prd.sh_result;
            prd.countEmitted = true;
            prd.done = false;
            prd.inside = false;
            prd.seed = seed;
            prd.depth = 0;
            prd.sh_coeff = 0.0 * prd.sh_coeff;

            float3 p;
            cosine_sample_hemisphere(z1, z2, p);
            float3 ray_direction = v1 * p.x + v2 * p.y + normal * p.z;
            ray_direction = normalize(ray_direction);

            float3 ray_origin = vertex_pos;

            for (;;) {
                Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
                rtTrace(top_object, ray, prd);
                if (prd.done) {
                    prd.sh_result += prd.sh_coeff;
                    break;
                }

                // RR: randomly reject some rays
                

                prd.depth++;
                prd.sh_result += prd.sh_coeff ;
                ray_origin = prd.origin;
                ray_direction = prd.direction;
				if (prd.depth == 1){
					break;
				}
            }


            li = li + prd.sh_result;
            // Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            // rtTrace(top_object, ray, prd);
        }
    }

    li = li / (resolution * resolution);
    // li = li * (2 * M_PIf / (resolution * resolution));

    output_buffer_1[launch_index] = make_float4(li[0], li[1], li[2], 0.0f);
    output_buffer_2[launch_index] = make_float4(li[3], li[4], li[5], 0.0f);
    output_buffer_3[launch_index] = make_float4(li[6], li[7], li[8], 0.0f);
}




