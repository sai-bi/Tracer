#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "helpers.h"
#include "path_tracer.h"
#include "random.h"
#include "utils.h"

using namespace optix;

struct PerRayData_radiance{
    float3 result;
    float importance;
    int depth;
};


struct PerRayData_pathtrace_shadow{
    bool inShadow;
};

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;


rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, current_prd, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

static __device__ inline float3 powf(float3 a, float exp)
{
  return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
}

// For miss program
rtDeclareVariable(float3,       bg_color, , );
rtDeclareVariable(float3,        emission_color, , );

// For envirnoment map
rtTextureSampler<float4, 2> envmap;

// For shadow ray
rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

// For vertex tracer
rtBuffer<MyVertex>  vertices;



RT_PROGRAM void exception(){
  output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


RT_PROGRAM void miss(){
  current_prd.result = bg_color;
}


RT_PROGRAM void shadow(){
  current_prd_shadow.inShadow = true;
  rtTerminateRay();
}


RT_PROGRAM void envmap_miss(){
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	current_prd.result = make_float3(tex2D(envmap, u, v));
}


RT_PROGRAM void one_bounce_diffuse_closest_hit(){
    float3 hit_point = ray.origin + t_hit * ray.direction;
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal               = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );
    float2 uv                     = make_float2(texcoord);

    float3 Kd = make_float3(tex2D(diffuse_map, uv.x, uv.y));
    float3 result = make_float3(0);

    // compute indirect bounce 
    if(prd.detph < 1){
        optix::Onb onb(ffnormal);
        unsigned int seed = rot_seed(rnd_seeds[launch_index], frame);
        const float inv_sqrt_samples = 1.0f / float(sqrt_num_samples);

        int nx = sqrt_num_samples;
        int ny = sqrt_num_samples;

        while(nx--){
            while(ny--){
                float u1 = (float(nx) + rnd(seed)) * inv_sqrt_samples;
                float u2 = (float(ny) + rnd(seed)) * inv_sqrt_samples;

                float3 dir;
                optix::cosine_sample_hemisphere(u1, u2, dir);
                onb.inverse_transform(dir);

                PerRayData_radiance radiance_prd;
                radiance_prd.importance = current_prd.importance * optix::luminance(Kd);
                radiance_prd.depth = current_prd.depth + 1;

                if(radiance_prd.importance > 0.001f){
                    optix::Ray radiance_ray = optix::make_Ray(hit_point, dir, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
                    rtTrace(top_object, radiance_ray, radiance_prd);
                    result += radiance_prd.result;
                }
            }
        }
        result *= (Kd) / ((float)(M_PIf * sqrt_num_samples * sqrt_num_samples));
    }

    current_prd.result = result;
}



RT_PROGRAM void VertexTracer(){
    int index = launch_index.x;
    float3 vertex_pos = vertices[index].vertex;
    float3 normal = vertices[index].normal;
    normal = normalize(normal);

    unsigned int seed = tea<16>(index, 1);
    unsigned int sample_num = samples_per_vertex;
    float3 result = make_float3(0.0f);

    float3 v1, v2;
    createONB(normal, v1, v2);
    float3 direct_light = make_float3(0.0);
    sample_num = 1000;


    do{
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.0f);
        prd.attenuation = make_float3(0.8f);
        prd.countEmitted = true;
        prd.done = false;
        prd.inside = false;
        prd.seed = seed;
        prd.depth = 0;

        // float z1 = (sample_num % 20 + 0.5) / 20;
        // float z2 = (sample_num / 20 + 0.5) / 20;
        float z1 = rnd(prd.seed);
        float z2 = rnd(prd.seed);

        float3 p;
        cosine_sample_hemisphere(z1, z2, p);
        float3 ray_direction = v1 * p.x  + v2 * p.y  + normal * p.z;

        ray_direction = normalize(ray_direction);
        float cos_theta = dot(ray_direction, normal);

        float3 ray_origin = vertex_pos;
        for(;;) {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);
            if(prd.done) {
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // RR: randomly reject some rays
            if(prd.depth >= rr_begin_depth){
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += (prd.result);
        seed = prd.seed;
    } while(--sample_num);


    result = result / 1000;

    float3 pixel_color = result;
    output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
}




RT_PROGRAM void InDirectRender(){
    unsigned int resolution = 10;
    int index = launch_index.x;
    float3 vertex_pos = vertices[index].vertex;
    float3 normal = vertices[index].normal;
    normal = normalize(normal);
    unsigned int seed = tea<16>(index, 1);

    float3 v1, v2;
    createONB(normal, v1, v2);
    Matrix3x3 li;
    li = 0.0 * li;

    for (unsigned int i = 0; i < resolution; i++){
        for (unsigned int j = 0; j < resolution; j++){
            float z1 = (i + 0.5) / (float)resolution;
            float z2 = (j + 0.5) / (float)resolution;
            
            PerRayData_pathtrace prd;
            prd.sh_result = 0.0f * prd.sh_result;
            prd.countEmitted = true;
            prd.done = false;
            prd.inside = false;
            prd.seed = seed;
            prd.depth = 0;
            prd.sh_coeff = 0.0 * prd.sh_coeff;

            float3 p;
            cosine_sample_hemisphere(z1, z2, p);
            float3 ray_direction = v1 * p.x + v2 * p.y + normal * p.z;
            ray_direction = normalize(ray_direction);

            float3 ray_origin = vertex_pos;

            for (;;) {
                Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
                rtTrace(top_object, ray, prd);
                if (prd.done) {
                    prd.sh_result += prd.sh_coeff;
                    break;
                }

                // RR: randomly reject some rays
                

                prd.depth++;
                prd.sh_result += prd.sh_coeff ;
                ray_origin = prd.origin;
                ray_direction = prd.direction;
				if (prd.depth == 1){
					break;
				}
            }


            li = li + prd.sh_result;
            // Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            // rtTrace(top_object, ray, prd);
        }
    }

    li = li / (resolution * resolution);
    // li = li * (2 * M_PIf / (resolution * resolution));

    output_buffer_1[launch_index] = make_float4(li[0], li[1], li[2], 0.0f);
    output_buffer_2[launch_index] = make_float4(li[3], li[4], li[5], 0.0f);
    output_buffer_3[launch_index] = make_float4(li[6], li[7], li[8], 0.0f);
}




