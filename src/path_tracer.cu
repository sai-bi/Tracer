#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "helpers.h"
#include "path_tracer.h"
#include "random.h"
#include "utils.h"

using namespace optix;

struct PerRayData_pathtrace
{
  float3 result;
  float3 radiance;
  float3 attenuation;
  float3 origin;
  float3 direction;
  unsigned int seed;
  int depth;
  int countEmitted;
  int done;
  int inside;

  // @sai bi
  Matrix3x3 sh_coeff;
  Matrix3x3 sh_result;
};

struct PerRayData_pathtrace_shadow
{
  bool inShadow;
};

// Scene wide
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );

// For camera
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtBuffer<float4, 2>              output_buffer;

rtBuffer<float4, 2> output_buffer_1;
rtBuffer<float4, 2> output_buffer_2;
rtBuffer<float4, 2> output_buffer_3;


rtBuffer<ParallelogramLight>     lights;

rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

static __device__ inline float3 powf(float3 a, float exp)
{
  return make_float3(powf(a.x, exp), powf(a.y, exp), powf(a.z, exp));
}

// For miss program
rtDeclareVariable(float3,       bg_color, , );
rtDeclareVariable(float3,        emission_color, , );

// For envirnoment map
rtTextureSampler<float4, 2> envmap;

// For shadow ray
rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

// For vertex tracer
rtBuffer<MyVertex>  vertices;

// 


RT_PROGRAM void exception(){
  output_buffer[launch_index] = make_float4(bad_color, 0.0f);
}


RT_PROGRAM void miss(){
  current_prd.radiance = bg_color;
  current_prd.done = true;
}


RT_PROGRAM void shadow(){
  current_prd_shadow.inShadow = true;
  rtTerminateRay();
}


RT_PROGRAM void envmap_miss(){
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	current_prd.radiance = make_float3(tex2D(envmap, u, v));
    current_prd.done = true;
}



RT_PROGRAM void one_bounce_diffuse_closest_hit(){

}



RT_PROGRAM void VertexTracer(){
    int index = launch_index.x;
    float3 vertex_pos = vertices[index].vertex;
    float3 normal = vertices[index].normal;
    normal = normalize(normal);

    unsigned int seed = tea<16>(index, 1);
    unsigned int sample_num = samples_per_vertex;
    float3 result = make_float3(0.0f);

    float3 v1, v2;
    createONB(normal, v1, v2);
    float3 direct_light = make_float3(0.0);
    sample_num = 1000;


    do{
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.0f);
        prd.attenuation = make_float3(0.8f);
        prd.countEmitted = true;
        prd.done = false;
        prd.inside = false;
        prd.seed = seed;
        prd.depth = 0;

        // float z1 = (sample_num % 20 + 0.5) / 20;
        // float z2 = (sample_num / 20 + 0.5) / 20;
        float z1 = rnd(prd.seed);
        float z2 = rnd(prd.seed);

        float3 p;
        cosine_sample_hemisphere(z1, z2, p);
        float3 ray_direction = v1 * p.x  + v2 * p.y  + normal * p.z;

        ray_direction = normalize(ray_direction);
        float cos_theta = dot(ray_direction, normal);

        float3 ray_origin = vertex_pos;
        for(;;) {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);
            if(prd.done) {
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // RR: randomly reject some rays
            if(prd.depth >= rr_begin_depth){
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += (prd.result);
        seed = prd.seed;
    } while(--sample_num);


    result = result / 1000;

    float3 pixel_color = result;
    output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
}




RT_PROGRAM void InDirectRender(){
    unsigned int resolution = 10;
    int index = launch_index.x;
    float3 vertex_pos = vertices[index].vertex;
    float3 normal = vertices[index].normal;
    normal = normalize(normal);
    unsigned int seed = tea<16>(index, 1);

    float3 v1, v2;
    createONB(normal, v1, v2);
    Matrix3x3 li;
    li = 0.0 * li;

    for (unsigned int i = 0; i < resolution; i++){
        for (unsigned int j = 0; j < resolution; j++){
            float z1 = (i + 0.5) / (float)resolution;
            float z2 = (j + 0.5) / (float)resolution;
            
            PerRayData_pathtrace prd;
            prd.sh_result = 0.0f * prd.sh_result;
            prd.countEmitted = true;
            prd.done = false;
            prd.inside = false;
            prd.seed = seed;
            prd.depth = 0;
            prd.sh_coeff = 0.0 * prd.sh_coeff;

            float3 p;
            cosine_sample_hemisphere(z1, z2, p);
            float3 ray_direction = v1 * p.x + v2 * p.y + normal * p.z;
            ray_direction = normalize(ray_direction);

            float3 ray_origin = vertex_pos;

            for (;;) {
                Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
                rtTrace(top_object, ray, prd);
                if (prd.done) {
                    prd.sh_result += prd.sh_coeff;
                    break;
                }

                // RR: randomly reject some rays
                

                prd.depth++;
                prd.sh_result += prd.sh_coeff ;
                ray_origin = prd.origin;
                ray_direction = prd.direction;
				if (prd.depth == 1){
					break;
				}
            }


            li = li + prd.sh_result;
            // Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            // rtTrace(top_object, ray, prd);
        }
    }

    li = li / (resolution * resolution);
    // li = li * (2 * M_PIf / (resolution * resolution));

    output_buffer_1[launch_index] = make_float4(li[0], li[1], li[2], 0.0f);
    output_buffer_2[launch_index] = make_float4(li[3], li[4], li[5], 0.0f);
    output_buffer_3[launch_index] = make_float4(li[6], li[7], li[8], 0.0f);
}




